#include <iostream>
#include <fstream>
#include <string>
#include <vector>

#include "FileManager.cuh"
#include "BooleanNet.cuh"
#include "InputParser.hpp"
#include "util.cuh"

#define MAX_N_IMP 25000000
#define MAX_N_SYM_IMP 1000000

using namespace std;

uint64_t round_div_up (uint64_t a, uint64_t b){
    return (a + b - 1)/b;
}

void launch_kernel (char * d_expr_values, uint64_t ngenes, int nsamples, BooleanNet * d_net, float statThresh, float pvalThresh, uint32_t * d_impl_len, impl * d_implications, uint32_t * d_symm_impl_len, symm_impl * d_symm_implications){
    int lws = 256;
    uint64_t gws = round_div_up(ngenes * ngenes, lws);
    cerr << "Launching kernel with " << gws << " work-groups and " << lws << " work-items per group" << " for " << ngenes*ngenes << " items" << endl;
    getImplication<<<gws, lws>>>(d_expr_values, ngenes, nsamples, d_net, statThresh, pvalThresh, d_impl_len, d_implications, d_symm_impl_len, d_symm_implications);
    hipError_t err = hipGetLastError();
    cuda_err_check(err, __FILE__, __LINE__);
}

void parse_arguments(int argc, char * argv[], string & expression_file, string & implication_file, float & statThresh, float & pvalThresh){
    InputParser pars(argc, argv);

    expression_file = "/home/luca/Development/IDM/Tesi/expr_discrete/expr_big.txt";
    implication_file = "/home/luca/Development/IDM/Tesi/impl.txt";

    statThresh = 3.0;
    pvalThresh = 0.1;

    if (pars.cmdOptionExists("-h")) {cerr << "Usage: " << argv[0] << " -i <expression_file> -s <statistic_threshold> -p <p-value_threshold> -o <implication_file>" << endl; exit(0);}
    if (pars.cmdOptionExists("-i")) expression_file = pars.getCmdOption("-i");  else cerr << "Warning: no expression file specified, using default: " << expression_file << endl;
    if (pars.cmdOptionExists("-s")) statThresh = stod(pars.getCmdOption("-s")); else cerr << "Warning: no statistic threshold specified, using default: " << statThresh << endl;
    if (pars.cmdOptionExists("-p")) pvalThresh = stod(pars.getCmdOption("-p")); else cerr << "Warning: no p-value threshold specified, using default: " << pvalThresh << endl;
    if (pars.cmdOptionExists("-o")) implication_file = pars.getCmdOption("-o"); else cerr << "Warning: no implication file specified, using default: " << implication_file << endl;
}

int main(int argc, char * argv[]){
    FileManager fm; hipError_t err;

    string expression_file, implication_file;
    float statThresh, pvalThresh;

    parse_arguments(argc, argv, expression_file, implication_file, statThresh, pvalThresh);

    vector<string> genes;
    char * expr_values;
    int n_rows, n_cols;
    fm.readFile(expression_file);
   
    genes = fm.getListGenes();
    expr_values = fm.getMatrix();
    n_rows = fm.getNumberOfRows();
    n_cols = fm.getNumberOfColumns();

    cerr << "Expression Matrix shape: " << n_rows << " x " << n_cols << endl;
    cerr << "Number of genes: " << genes.size() << endl;

    // cuda Malloc --------------------------------------------

    BooleanNet net;
    BooleanNet * d_net;
    err = hipMalloc(&d_net, sizeof(BooleanNet));
    cuda_err_check(err, __FILE__, __LINE__);

    char * d_expr_values;
    err = hipMalloc(&d_expr_values, sizeof(char) * n_rows * n_cols);
    cuda_err_check(err, __FILE__, __LINE__);

    uint32_t impl_len;
    uint32_t * d_impl_len;
    err = hipMalloc(&d_impl_len, sizeof(uint32_t));
    cuda_err_check(err, __FILE__, __LINE__);

    impl * d_implications;
    err = hipMalloc(&d_implications, sizeof(impl) * MAX_N_IMP);
    cuda_err_check(err, __FILE__, __LINE__);

    uint32_t symm_impl_len;
    uint32_t * d_symm_impl_len;
    err = hipMalloc(&d_symm_impl_len, sizeof(uint32_t));
    cuda_err_check(err, __FILE__, __LINE__);

    symm_impl * d_symm_implications;
    err = hipMalloc(&d_symm_implications, sizeof(symm_impl) * MAX_N_SYM_IMP);
    cuda_err_check(err, __FILE__, __LINE__);

    // cuda Memcpy --------------------------------------------

    err = hipMemcpy(d_expr_values, expr_values, sizeof(char) * n_rows * n_cols, hipMemcpyHostToDevice);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(d_net, &net, sizeof(BooleanNet), hipMemcpyHostToDevice);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemset(d_impl_len, 0, sizeof(uint32_t));
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemset(d_symm_impl_len, 0, sizeof(uint32_t));
    cuda_err_check(err, __FILE__, __LINE__);

    cerr << "Instantiated Implications Matrix of size: " << genes.size() * genes.size() /** 4 * 5*/ << endl;

    // Launch kernel ------------------------------------------

    launch_kernel(d_expr_values, n_rows, n_cols, d_net, statThresh, pvalThresh, d_impl_len, d_implications, d_symm_impl_len, d_symm_implications);

    hipDeviceSynchronize();

    // Copy back results --------------------------------------

    cerr << "Kernel execution completed" << endl;

    err = hipMemcpy(&impl_len, d_impl_len, sizeof(uint32_t), hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    cerr << "Number of implications: " << impl_len << endl;

    err = hipMemcpy(&symm_impl_len, d_symm_impl_len, sizeof(uint32_t), hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    cerr << "Number of symmetric implications: " << symm_impl_len << endl;

    if (impl_len > MAX_N_IMP || symm_impl_len > MAX_N_SYM_IMP){
        cerr << "Error! Too many implications!" << endl;
        exit(1);
    }

    impl * implications;
    err = hipHostMalloc(&implications, sizeof(impl) * impl_len);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(implications, d_implications, sizeof(impl) * impl_len, hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    symm_impl * symm_implications;
    err = hipHostMalloc(&symm_implications, sizeof(symm_impl) * symm_impl_len);

    err = hipMemcpy(symm_implications, d_symm_implications, sizeof(symm_impl) * symm_impl_len, hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    // Print results ------------------------------------------

    fm.writeImplications(implication_file, genes, impl_len, implications, symm_impl_len, symm_implications);

    // Free memory --------------------------------------------

    err = hipFree(d_expr_values);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipFree(d_net);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipFree(d_impl_len);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipFree(d_implications);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipFree(d_symm_impl_len);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipFree(d_symm_implications);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostFree(implications);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostFree(symm_implications);
    cuda_err_check(err, __FILE__, __LINE__);

    return 0;
}
