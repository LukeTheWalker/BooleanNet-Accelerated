#include <iostream>
#include <fstream>
#include <string>
#include <vector>

#include "FileManager.cuh"
#include "BooleanNet.cuh"
#include "InputParser.hpp"
#include "util.cuh"

using namespace std;

uint32_t round_div_up (uint32_t a, uint32_t b){
    return (a + b - 1)/b;
}

void StepMinerCompression (char * expression_values_char, uint32_t *expr_values, uint32_t * zero_flags, uint32_t ngenes, int nsamples){
    int nbits = sizeof(*zero_flags) * 8;
    int nslots = round_div_up(nsamples, nbits);
    for (uint32_t i = 0; i < ngenes; i++){
        uint32_t * zero_flags_row = zero_flags + i * nslots;
        uint32_t * discretizedValues_row = expr_values + i * nslots;
        for (int j = 0; j < nsamples; j++){
            int byte_to_access = j / nbits;
            int bit_to_access = j % nbits;
            if (expression_values_char[i * nsamples + j] == -1){
                BIT_SET(*(zero_flags_row + byte_to_access), bit_to_access);
                BIT_CLEAR(*(discretizedValues_row + byte_to_access), bit_to_access);
            }
            else if (expression_values_char[i * nsamples + j] == 1){
                BIT_SET(*(zero_flags_row + byte_to_access), bit_to_access);
                BIT_SET(*(discretizedValues_row + byte_to_access), bit_to_access);

            }
            else {
                BIT_CLEAR(*(zero_flags_row + byte_to_access), bit_to_access);
                BIT_CLEAR(*(discretizedValues_row + byte_to_access), bit_to_access);
            }
        }
    }
}


void launch_kernel (uint32_t *d_expr_values, uint32_t * d_zero_flags, uint32_t ngenes, int nsamples, float statThresh, float pvalThresh, uint32_t * d_impl_len, impl * d_implications, uint32_t * d_symm_impl_len, symm_impl * d_symm_implications, bool full_precision){
    // int lws = 256;
    int nbits = sizeof(*d_zero_flags) * 8;
    int nslots = round_div_up(nsamples, nbits);
    dim3 lws(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gws(round_div_up(ngenes, lws.x), round_div_up(ngenes, lws.y), 1);
    cerr << "Launching kernel with " << gws.x << " x " << gws.y << " work-groups and " << lws.x << " x " << lws.y << " work-items per group" << endl;

    hipError_t err;
    hipEvent_t start, stop;
    err = hipEventCreate(&start); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventCreate(&stop); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventRecord(start); cuda_err_check(err, __FILE__, __LINE__);

    if (full_precision)
    BooleanNet::getImplication<double><<<gws, lws>>>(d_expr_values, d_zero_flags, ngenes, nsamples, statThresh, pvalThresh, d_impl_len, d_implications, d_symm_impl_len, d_symm_implications);
    else 
    BooleanNet::getImplication<float><<<gws, lws>>>(d_expr_values, d_zero_flags, ngenes, nsamples, statThresh, pvalThresh, d_impl_len, d_implications, d_symm_impl_len, d_symm_implications);
    err = hipEventRecord(stop); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventSynchronize(stop); cuda_err_check(err, __FILE__, __LINE__);

    err = hipGetLastError(); cuda_err_check(err, __FILE__, __LINE__);
    err = hipDeviceSynchronize(); cuda_err_check(err, __FILE__, __LINE__);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cerr << "Kernel execution time: " << milliseconds << " ms" << endl;

    err = hipEventDestroy(start); cuda_err_check(err, __FILE__, __LINE__);
    err = hipEventDestroy(stop); cuda_err_check(err, __FILE__, __LINE__);
}

void parse_arguments(int argc, char * argv[], string & expression_file, string & implication_file, float & statThresh, float & pvalThresh, bool & full_precision){
    InputParser pars(argc, argv);

    expression_file = "/home/luca/Development/IDM/Tesi/expr_discrete/expr_big.txt";
    implication_file = "/home/luca/Development/IDM/Tesi/impl.txt";

    statThresh = 3.0;
    pvalThresh = 0.1;

    full_precision = false;

    if (pars.cmdOptionExists("-h")) {cerr << "Usage: " << argv[0] << " -i <expression_file> -s <statistic_threshold> -p <p-value_threshold> -o <implication_file>" << endl; exit(0);}
    if (pars.cmdOptionExists("-i")) expression_file = pars.getCmdOption("-i");  else cerr << "Warning: no expression file specified, using default: " << expression_file << endl;
    if (pars.cmdOptionExists("-s")) statThresh = stod(pars.getCmdOption("-s")); else cerr << "Warning: no statistic threshold specified, using default: " << statThresh << endl;
    if (pars.cmdOptionExists("-p")) pvalThresh = stod(pars.getCmdOption("-p")); else cerr << "Warning: no p-value threshold specified, using default: " << pvalThresh << endl;
    if (pars.cmdOptionExists("-o")) implication_file = pars.getCmdOption("-o"); else cerr << "Warning: no implication file specified, using default: " << implication_file << endl;
    if (pars.cmdOptionExists("-fp64")) full_precision = true; else cerr << "Warning: using compressed representation" << endl;
}

int main(int argc, char * argv[]){
    FileManager fm; hipError_t err;

    string expression_file, implication_file;
    float statThresh, pvalThresh;

    bool full_precision;

    parse_arguments(argc, argv, expression_file, implication_file, statThresh, pvalThresh, full_precision);

    vector<string> genes;
    char * expr_values_char;
    int n_rows, n_cols;
    fm.readFile(expression_file);
   
    genes = fm.getListGenes();
    expr_values_char = fm.getMatrix();
    n_rows = fm.getNumberOfRows();
    n_cols = fm.getNumberOfColumns();

    uint32_t * expr_values;
    uint32_t * zero_flags;
    err = hipHostMalloc(&expr_values, sizeof(uint32_t) * n_rows * n_cols); cuda_err_check(err, __FILE__, __LINE__);
    err = hipHostMalloc(&zero_flags, sizeof(uint32_t) * n_rows * n_cols); cuda_err_check(err, __FILE__, __LINE__);

    StepMinerCompression(expr_values_char, expr_values, zero_flags, n_rows, n_cols);

    int nbits = sizeof(*zero_flags) * 8;
    int nslots = round_div_up(n_cols, nbits);

    cerr << "Expression Matrix shape: " << n_rows << " x " << n_cols << endl;
    cerr << "Number of genes: " << genes.size() << endl;

    // cuda Malloc --------------------------------------------

    uint32_t impl_len;
    uint32_t * d_impl_len;
    err = hipMalloc(&d_impl_len, sizeof(uint32_t)); cuda_err_check(err, __FILE__, __LINE__);

    impl * d_implications;
    err = hipMalloc(&d_implications, sizeof(impl) * MAX_N_IMP); cuda_err_check(err, __FILE__, __LINE__);
    cerr << "Max number of implications acceptable: " << MAX_N_IMP << endl;

    uint32_t symm_impl_len;
    uint32_t * d_symm_impl_len;
    err = hipMalloc(&d_symm_impl_len, sizeof(uint32_t)); cuda_err_check(err, __FILE__, __LINE__);

    symm_impl * d_symm_implications;
    err = hipMalloc(&d_symm_implications, sizeof(symm_impl) * MAX_N_SYM_IMP); cuda_err_check(err, __FILE__, __LINE__);
    cerr << "Max number of symmetric implications acceptable: " << MAX_N_SYM_IMP << endl;

    uint32_t * d_zero_flags;
    err = hipMalloc(&d_zero_flags, sizeof(uint32_t) * n_rows * nslots); cuda_err_check(err, __FILE__, __LINE__);

    uint32_t * d_expr_values;
    err = hipMalloc(&d_expr_values, sizeof(uint32_t) * n_rows * nslots); cuda_err_check(err, __FILE__, __LINE__);


    // cuda Memcpy --------------------------------------------

    err = hipMemset(d_impl_len, 0, sizeof(uint32_t)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemset(d_symm_impl_len, 0, sizeof(uint32_t)); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_zero_flags, zero_flags, sizeof(uint32_t) * n_rows * nslots, hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);
    err = hipMemcpy(d_expr_values, expr_values, sizeof(uint32_t) * n_rows * nslots, hipMemcpyHostToDevice); cuda_err_check(err, __FILE__, __LINE__);

    // Launch kernel ------------------------------------------

    launch_kernel(d_expr_values, d_zero_flags, n_rows, n_cols, statThresh, pvalThresh, d_impl_len, d_implications, d_symm_impl_len, d_symm_implications, full_precision);

    hipDeviceSynchronize();

    // Copy back results --------------------------------------

    cerr << "Kernel execution completed" << endl;

    err = hipMemcpy(&impl_len, d_impl_len, sizeof(uint32_t), hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    cerr << "Number of asymmetric implications: " << impl_len << endl;

    err = hipMemcpy(&symm_impl_len, d_symm_impl_len, sizeof(uint32_t), hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    cerr << "Number of symmetric implications:  " << symm_impl_len << endl;

    if (impl_len > MAX_N_IMP || symm_impl_len > MAX_N_SYM_IMP){
        cerr << "Error! Too many implications!" << endl;
        exit(1);
    }

    impl * implications;
    err = hipHostMalloc(&implications, sizeof(impl) * impl_len);
    cuda_err_check(err, __FILE__, __LINE__);

    err = hipMemcpy(implications, d_implications, sizeof(impl) * impl_len, hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    symm_impl * symm_implications;
    err = hipHostMalloc(&symm_implications, sizeof(symm_impl) * symm_impl_len);

    err = hipMemcpy(symm_implications, d_symm_implications, sizeof(symm_impl) * symm_impl_len, hipMemcpyDeviceToHost);
    cuda_err_check(err, __FILE__, __LINE__);

    // Print results ------------------------------------------

    fm.writeImplications(implication_file, genes, impl_len, implications, symm_impl_len, symm_implications);

    // Free memory --------------------------------------------

    err = hipFree(d_expr_values); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_impl_len); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_implications); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_symm_impl_len); cuda_err_check(err, __FILE__, __LINE__);
    err = hipFree(d_symm_implications); cuda_err_check(err, __FILE__, __LINE__);

    err = hipHostFree(implications); cuda_err_check(err, __FILE__, __LINE__);
    err = hipHostFree(symm_implications); cuda_err_check(err, __FILE__, __LINE__);

    return 0;
}
