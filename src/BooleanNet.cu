#include "hip/hip_runtime.h"
#include "BooleanNet.cuh"

// NOTE: strict mapping of this function for impl_type in [0,3].
__device__ char get_inverse_implication(char impl_type){
    // Not using swith case because of problems with some gpu's and compiler, but still better than 4 if-else
    return (3 - impl_type) - (impl_type == 1) + (impl_type == 2);
}

__host__ void BooleanNet::get_all_implications(std::vector<std::string> genes, char* expr_values, int nsamples, float statThresh, float pvalThresh, float * implication_matrix){
#if 1
    int gene1, gene2;
    int n_first_low, n_first_high, n_second_high, n_second_low, n_total;
    float statistic, pval;
    int i = 0;
    for (gene1 = 0; gene1 < genes.size(); gene1++){
        for (gene2 = 0; gene2 < genes.size(); gene2++){
            if (gene1 != gene2){
                int quadrant_counts[4];
                getQuadrantCounts(gene1, gene2, expr_values, nsamples, quadrant_counts);

                // for (int i = 0; i < 4; i++){
                //     if (i == 2) printf("\n");
                //     printf("%d\t", quadrant_counts[i]);
                // }
                // printf("\n");

                n_first_low = quadrant_counts[0] + quadrant_counts[1];
                n_first_high = quadrant_counts[2] + quadrant_counts[3];
                n_second_high = quadrant_counts[1] + quadrant_counts[3];
                n_second_low = quadrant_counts[0] + quadrant_counts[2];

                n_total = n_first_low + n_first_high;

                for (char impl_type = 0; impl_type < 4; impl_type++){
                    getSingleImplication(quadrant_counts, n_total, n_first_low, n_first_high, n_second_low, n_second_high, impl_type, &statistic, &pval);
                    if (statistic > statThresh && pval < pvalThresh){
                        // implication_matrix[i] = gene1;
                        // implication_matrix[i+1] = gene2;
                        // implication_matrix[i+2] = impl_type;
                        // implication_matrix[i+3] = statistic;
                        // implication_matrix[i+4] = pval;
                        printf("%s\t%s\t%d\t%f\t%f\t\n", genes[gene1].c_str(), genes[gene2].c_str(), impl_type, statistic, pval);
                    }
                    i += 5;
                }
            }
        }
    }
#endif
}

__host__ __device__ void BooleanNet::getQuadrantCounts(int gene1, int gene2, char* expr_values, int nsamples, int* quadrant_counts){
    quadrant_counts[0] = quadrant_counts[1] = quadrant_counts[2] = quadrant_counts[3] = 0;
    int g1_ns = gene1 * nsamples;
    int g2_ns = gene2 * nsamples;
    for (int i = 0; i < nsamples; i++){
        bool k1p = g1_ns + i == 1;
        bool k1n = g1_ns + i == -1;
        bool k2p = g2_ns + i == 1;
        bool k2n = g2_ns + i == -1;
        quadrant_counts[0] += k2n && k1n;
        quadrant_counts[1] += k2p && k1n;
        quadrant_counts[2] += k2n && k1p;
        quadrant_counts[3] += k2p && k1p;
    }
}

__host__ __device__ void BooleanNet::getSingleImplication(int* quadrant_counts, int n_total, int n_first_low, int n_first_high, int n_second_low, int n_second_high, char impl_type, float* statistic, float* pval){
    if (is_zero(n_first_low, n_first_high, n_second_low, n_second_high, impl_type)){
        *statistic = 0.0;
        *pval = 1.0;
        return;
    }

    int n1 = (impl_type > 1) * n_first_high + (impl_type <= 1) * n_first_low;
    int n2 = (impl_type & 1) * n_second_low + (1 - (impl_type & 1)) * n_second_high;
    int np = n1 * n2;
    int q_index = impl_type ^ 1;
    double n_expected = (double)np / n_total;
    *statistic = (n_expected - quadrant_counts[q_index] / sqrt(n_expected));
    *pval = ((double)(n1 + n2)) / (2 * np) * quadrant_counts[q_index];

}
__host__ __device__ char BooleanNet::is_zero(int n_first_low, int n_first_high, int n_second_low, int n_second_high, char impl_type){
#ifdef DEBUG
    if (impl_type & 0xfffffffc) printf("Invalid impl_type in is_zero\n");
#endif

    // For an explanation of the below method, see also:
    // https://en.wikipedia.org/wiki/Karnaugh_map
    // https://en.wikipedia.org/wiki/Quine%E2%80%93McCluskey_algorithm

    bool ih = impl_type & 2;        // a
    bool il = impl_type & 1;        // b
    bool n1l = n_first_low > 0;     // c
    bool n1h = n_first_high > 0;    // d
    bool n2l = n_second_low > 0;    // e
    bool n2h = n_second_high > 0;   // f

    // SOP form
    // Calculated at: http://www.32x8.com/qmm6_____A-B-C-D-E-F_____m_0-1-2-3-4-5-6-7-8-10-11-12-13-14-15-16-17-18-19-20-21-22-23-24-25-27-28-29-30-31-32-33-34-35-36-38-39-40-41-42-43-44-45-46-47-48-49-50-51-52-53-55-56-57-58-59-60-61-62-63___________option-4_____899788965371824592779
    return  (!ih && !n1l) || (!il && !n2h) || (n2l && n2h) ||
            (n1l && n1h)  || (il && !n2h)  || (ih && !n1h);
}

__global__ void getImplication(char * expr_values, uint64_t ngenes, int nsamples, BooleanNet * net, float statThresh, float pvalThresh, uint32_t * impl_len, impl * d_implications, uint32_t * d_symm_impl_len, symm_impl * d_symm_implications){
    uint64_t gi = (uint64_t) blockIdx.x * (uint64_t) blockDim.x + (uint64_t) threadIdx.x;

    // uint64_t gene1 = gi / ngenes;
    // uint64_t gene2 = gi % ngenes;

    uint64_t gene1 = ngenes - 2 - floor(sqrt((double)-8*gi + 4*ngenes*(ngenes-1)-7)/2.0 - 0.5);
    uint64_t gene2 = gi + gene1 + 1 - ngenes*(ngenes-1)/2 + (ngenes-gene1)*((ngenes-gene1)-1)/2;

    uint64_t nels = (ngenes * (ngenes - 1)) / 2;

    // if (gi % (nels / 100) == 0)
    //     printf("Processed %ld%% of %ld total genes, index: %ld\n", gi / (nels / 100), nels, gi);
    
    if (gene1 == gene2 || gi >= nels){
        return;
    }

    int n_first_low, n_first_high, n_second_high, n_second_low, n_total;
    float all_statistic[4], all_pval[4];

    int quadrant_counts[4];
    net->getQuadrantCounts(gene1, gene2, expr_values, nsamples, quadrant_counts);

    n_first_low = quadrant_counts[0] + quadrant_counts[1];
    n_first_high = quadrant_counts[2] + quadrant_counts[3];
    n_second_high = quadrant_counts[1] + quadrant_counts[3];
    n_second_low = quadrant_counts[0] + quadrant_counts[2];

    n_total = n_first_low + n_first_high;

    for (char impl_type = 0; impl_type < 4; impl_type++){
        float * statistic = all_statistic + impl_type;
        float * pval = all_pval + impl_type;
        net->getSingleImplication(quadrant_counts, n_total, n_first_low, n_first_high, n_second_low, n_second_high, impl_type, statistic, pval);
        if (*statistic >= statThresh && *pval <= pvalThresh){
            int idx = atomicAdd(impl_len, 1);
            d_implications[idx] = {(int)gene1, (int)gene2, impl_type, *statistic, *pval};
            idx = atomicAdd(impl_len, 1);
            d_implications[idx] = {(int)gene2, (int)gene1, get_inverse_implication(impl_type), *statistic, *pval};
        }
    }
    if (all_statistic[0] >= statThresh && all_pval[0] <= pvalThresh && all_statistic[3] >= statThresh && all_pval[3] <= pvalThresh){
        int idx = atomicAdd(d_symm_impl_len, 1);
        d_symm_implications[idx] = {(int)gene1, (int)gene2, 4, all_statistic[0], all_statistic[3], all_pval[0], all_pval[3]};
        idx = atomicAdd(d_symm_impl_len, 1);
        d_symm_implications[idx] = {(int)gene2, (int)gene1, 4, all_statistic[3], all_statistic[0], all_pval[3], all_pval[0]};
    }
    else if (all_statistic[1] >= statThresh && all_pval[1] <= pvalThresh && all_statistic[2] >= statThresh && all_pval[2] <= pvalThresh){
        int idx = atomicAdd(d_symm_impl_len, 1);
        d_symm_implications[idx] = {(int)gene1, (int)gene2, 5, all_statistic[1], all_statistic[2], all_pval[1], all_pval[2]};
        idx = atomicAdd(d_symm_impl_len, 1);
        d_symm_implications[idx] = {(int)gene2, (int)gene1, 5, all_statistic[2], all_statistic[1], all_pval[2], all_pval[1]};
    }
}

