#include "hip/hip_runtime.h"
#include "BooleanNet.cuh"

__device__ char get_inverse_implication(char impl_type){
    if (impl_type == 0){
        return 3;
    }
    else if (impl_type == 1){
        return 1;
    }
    else if (impl_type == 2){
        return 2;
    }
    else if (impl_type == 3){
        return 0;
    }
}

__device__ void getQuadrantCounts(int gene1, int gene2, char* expr_values, int nsamples, int* quadrant_counts){
    for (int i = 0; i < 4; i++){
        quadrant_counts[i] = 0;
    }
    for (int i = 0; i < nsamples; i++){
        if (expr_values[gene1 * nsamples + i] == -1){
            if (expr_values[gene2 * nsamples + i] == -1){
                quadrant_counts[0]++;
            }
            else if (expr_values[gene2 * nsamples + i] == 1){
                quadrant_counts[1]++;
            }
        }
        else if (expr_values[gene1 * nsamples + i] == 1){
            if (expr_values[gene2 * nsamples + i] == -1){
                quadrant_counts[2]++;
            }
            else if (expr_values[gene2 * nsamples + i] == 1){
                quadrant_counts[3]++;
            }
        }
    }
}

__device__ char is_zero(int n_first_low, int n_first_high, int n_second_low, int n_second_high, char impl_type){
    if (impl_type == 0){
        if (n_first_low > 0 && n_second_high > 0)
            return 0;
    }
    else if (impl_type == 1){
        if (n_first_low > 0 && n_second_low > 0)
            return 0;
    }
    else if (impl_type == 2){
        if (n_first_high > 0 && n_second_high > 0)
            return 0;
    }
    else if (impl_type == 3){
        if (n_first_high > 0 && n_second_low > 0)
            return 0;
    }
    else {
        printf("Invalid impl_type in is_zero\n");
    }
    return 1;
}


__device__ void getSingleImplication(int* quadrant_counts, int n_total, int n_first_low, int n_first_high, int n_second_low, int n_second_high, char impl_type, float* statistic, float* pval){
    if (is_zero(n_first_low, n_first_high, n_second_low, n_second_high, impl_type)){
        *statistic = 0.0;
        *pval = 1.0;
        return;
    }

    if (impl_type == 0){
        double n_expected = (double)(n_first_low * n_second_high) / n_total;
        *statistic = (n_expected - quadrant_counts[1]) / sqrt(n_expected);
        *pval = ((((double)quadrant_counts[1] / n_first_low) + ((double)quadrant_counts[1] / n_second_high)) / 2);
    }
    else if (impl_type == 1){
        double n_expected = (double)(n_first_low * n_second_low) / n_total;
        *statistic = (n_expected - quadrant_counts[0]) / sqrt(n_expected);
        *pval = ((((double)quadrant_counts[0] / n_first_low) + ((double)quadrant_counts[0] / n_second_low)) / 2);
    }
    else if (impl_type == 2){
        double n_expected = (double)(n_first_high * n_second_high) / n_total;
        *statistic = (n_expected - quadrant_counts[3]) / sqrt(n_expected);
        *pval = ((((double)quadrant_counts[3] / n_first_high) + ((double)quadrant_counts[3] / n_second_high)) / 2);
    }
    else if (impl_type == 3){
        double n_expected = (double)(n_first_high * n_second_low) / n_total;
        *statistic = (n_expected - quadrant_counts[2]) / sqrt(n_expected);
        *pval = ((((double)quadrant_counts[2] / n_first_high) + ((double)quadrant_counts[2] / n_second_low)) / 2);
    }
}

__global__ void BooleanNet::getImplication(char * expr_values, uint64_t ngenes, int nsamples, float statThresh, float pvalThresh, uint32_t * impl_len, impl * d_implications, uint32_t * d_symm_impl_len, symm_impl * d_symm_implications){
    uint64_t gi = (uint64_t) blockIdx.x * (uint64_t) blockDim.x + (uint64_t) threadIdx.x;

    uint64_t gene1 = ngenes - 2 - floor(sqrt((double)-8*gi + 4*ngenes*(ngenes-1)-7)/2.0 - 0.5);
    uint64_t gene2 = gi + gene1 + 1 - ngenes*(ngenes-1)/2 + (ngenes-gene1)*((ngenes-gene1)-1)/2;

    uint64_t nels = (ngenes * (ngenes - 1)) / 2;
    
    if (gene1 == gene2 || gi >= nels){
        return;
    }

    int n_first_low, n_first_high, n_second_high, n_second_low, n_total;
    float all_statistic[4], all_pval[4];

    int quadrant_counts[4];
    getQuadrantCounts(gene1, gene2, expr_values, nsamples, quadrant_counts);

    n_first_low = quadrant_counts[0] + quadrant_counts[1];
    n_first_high = quadrant_counts[2] + quadrant_counts[3];
    n_second_high = quadrant_counts[1] + quadrant_counts[3];
    n_second_low = quadrant_counts[0] + quadrant_counts[2];

    n_total = n_first_low + n_first_high;

    for (char impl_type = 0; impl_type < 4; impl_type++){
        float * statistic = all_statistic + impl_type;
        float * pval = all_pval + impl_type;
        getSingleImplication(quadrant_counts, n_total, n_first_low, n_first_high, n_second_low, n_second_high, impl_type, statistic, pval);
        if (*statistic >= statThresh && *pval <= pvalThresh){
            int idx = atomicAdd(impl_len, 2);
            d_implications[idx] = {(int)gene1, (int)gene2, impl_type, *statistic, *pval};
            d_implications[idx + 1] = {(int)gene2, (int)gene1, get_inverse_implication(impl_type), *statistic, *pval};
        }
    }
    if (all_statistic[0] >= statThresh && all_pval[0] <= pvalThresh && all_statistic[3] >= statThresh && all_pval[3] <= pvalThresh){
        int idx = atomicAdd(d_symm_impl_len, 2);
        d_symm_implications[idx] = {(int)gene1, (int)gene2, 4, all_statistic[0], all_statistic[3], all_pval[0], all_pval[3]};
        d_symm_implications[idx + 1] = {(int)gene2, (int)gene1, 4, all_statistic[3], all_statistic[0], all_pval[3], all_pval[0]};
    }
    else if (all_statistic[1] >= statThresh && all_pval[1] <= pvalThresh && all_statistic[2] >= statThresh && all_pval[2] <= pvalThresh){
        int idx = atomicAdd(d_symm_impl_len, 2);
        d_symm_implications[idx] = {(int)gene1, (int)gene2, 5, all_statistic[1], all_statistic[2], all_pval[1], all_pval[2]};
        d_symm_implications[idx + 1] = {(int)gene2, (int)gene1, 5, all_statistic[2], all_statistic[1], all_pval[2], all_pval[1]};
    }
}

