#include "hip/hip_runtime.h"
#include "BooleanNet.cuh"

__host__ void BooleanNet::get_all_implications(std::vector<std::string> genes, char* expr_values, int nsamples, float statThresh, float pvalThresh, float * implication_matrix){
#if 1
    int gene1, gene2;
    int n_first_low, n_first_high, n_second_high, n_second_low, n_total;
    float statistic, pval;
    int i = 0;
    for (gene1 = 0; gene1 < genes.size(); gene1++){
        for (gene2 = 0; gene2 < genes.size(); gene2++){
            if (gene1 != gene2){
                int quadrant_counts[4];
                getQuadrantCounts(gene1, gene2, expr_values, nsamples, quadrant_counts);

                // for (int i = 0; i < 4; i++){
                //     if (i == 2) printf("\n");
                //     printf("%d\t", quadrant_counts[i]);
                // }
                // printf("\n");

                n_first_low = quadrant_counts[0] + quadrant_counts[1];
                n_first_high = quadrant_counts[2] + quadrant_counts[3];
                n_second_high = quadrant_counts[1] + quadrant_counts[3];
                n_second_low = quadrant_counts[0] + quadrant_counts[2];

                n_total = n_first_low + n_first_high;

                for (char impl_type = 0; impl_type < 4; impl_type++){
                    getSingleImplication(quadrant_counts, n_total, n_first_low, n_first_high, n_second_low, n_second_high, impl_type, &statistic, &pval);
                    if (statistic > statThresh && pval < pvalThresh){
                        // implication_matrix[i] = gene1;
                        // implication_matrix[i+1] = gene2;
                        // implication_matrix[i+2] = impl_type;
                        // implication_matrix[i+3] = statistic;
                        // implication_matrix[i+4] = pval;
                        printf("%s\t%s\t%d\t%f\t%f\t\n", genes[gene1].c_str(), genes[gene2].c_str(), impl_type, statistic, pval);
                    }
                    i += 5;
                }
            }
        }
    }
#endif
}

__host__ __device__ void BooleanNet::getQuadrantCounts(int gene1, int gene2, char* expr_values, int nsamples, int* quadrant_counts){
    for (int i = 0; i < 4; i++){
        quadrant_counts[i] = 0;
    }
    // for (int i = 0; i < nsamples; i++){
    //     printf("%d\t", expr_values[gene1 * nsamples + i]);
    // }
    // printf("\n");
    // for (int i = 0; i < nsamples; i++){
    //     printf("%d\t", expr_values[gene2 * nsamples + i]);
    // }
    // printf("\n");
    for (int i = 0; i < nsamples; i++){
        if (expr_values[gene1 * nsamples + i] == -1){
            if (expr_values[gene2 * nsamples + i] == -1){
                quadrant_counts[0]++;
            }
            else if (expr_values[gene2 * nsamples + i] == 1){
                quadrant_counts[1]++;
            }
        }
        else if (expr_values[gene1 * nsamples + i] == 1){
            if (expr_values[gene2 * nsamples + i] == -1){
                quadrant_counts[2]++;
            }
            else if (expr_values[gene2 * nsamples + i] == 1){
                quadrant_counts[3]++;
            }
        }
    }
}

__host__ __device__ void BooleanNet::getSingleImplication(int* quadrant_counts, int n_total, int n_first_low, int n_first_high, int n_second_low, int n_second_high, char impl_type, float* statistic, float* pval){
    if (is_zero(n_first_low, n_first_high, n_second_low, n_second_high, impl_type)){
        *statistic = 0.0;
        *pval = 1.0;
        return;
    }

    if (impl_type == 0){
        double n_expected = (double)(n_first_low * n_second_high) / n_total;
        *statistic = (n_expected - quadrant_counts[1]) / sqrt(n_expected);
        *pval = ((((double)quadrant_counts[1] / n_first_low) + ((double)quadrant_counts[1] / n_second_high)) / 2);
    }
    else if (impl_type == 1){
        double n_expected = (double)(n_first_low * n_second_low) / n_total;
        *statistic = (n_expected - quadrant_counts[0]) / sqrt(n_expected);
        *pval = ((((double)quadrant_counts[0] / n_first_low) + ((double)quadrant_counts[0] / n_second_low)) / 2);
    }
    else if (impl_type == 2){
        double n_expected = (double)(n_first_high * n_second_high) / n_total;
        *statistic = (n_expected - quadrant_counts[3]) / sqrt(n_expected);
        *pval = ((((double)quadrant_counts[3] / n_first_high) + ((double)quadrant_counts[3] / n_second_high)) / 2);
    }
    else if (impl_type == 3){
        double n_expected = (double)(n_first_high * n_second_low) / n_total;
        *statistic = (n_expected - quadrant_counts[2]) / sqrt(n_expected);
        *pval = ((((double)quadrant_counts[2] / n_first_high) + ((double)quadrant_counts[2] / n_second_low)) / 2);
    }
}
__host__ __device__ char BooleanNet::is_zero(int n_first_low, int n_first_high, int n_second_low, int n_second_high, char impl_type){
    if (impl_type == 0){
        if (n_first_low > 0 && n_second_high > 0)
            return 0;
    }
    else if (impl_type == 1){
        if (n_first_low > 0 && n_second_low > 0)
            return 0;
    }
    else if (impl_type == 2){
        if (n_first_high > 0 && n_second_high > 0)
            return 0;
    }
    else if (impl_type == 3){
        if (n_first_high > 0 && n_second_low > 0)
            return 0;
    }
    else {
        printf("Invalid impl_type in is_zero\n");
    }
    return 1;
}

__global__ void getImplication(char * expr_values, uint64_t ngenes, int nsamples, BooleanNet * net, float statThresh, float pvalThresh){
    uint64_t gi = (uint64_t) blockIdx.x * (uint64_t) blockDim.x + (uint64_t) threadIdx.x;

    uint64_t gene1 = gi / ngenes;
    uint64_t gene2 = gi % ngenes;

    uint64_t nels = ngenes * ngenes;

    // if (gi % (nels / 100) == 0)
    //     printf("Processed %ld%% of %ld total genes, index: %ld\n", gi / (nels / 100), nels, gi);
    

    if (gene1 == gene2 || gi >= ngenes * ngenes){
        return;
    }

    int n_first_low, n_first_high, n_second_high, n_second_low, n_total;
    float statistic, pval;

    int quadrant_counts[4];
    net->getQuadrantCounts(gene1, gene2, expr_values, nsamples, quadrant_counts);

    n_first_low = quadrant_counts[0] + quadrant_counts[1];
    n_first_high = quadrant_counts[2] + quadrant_counts[3];
    n_second_high = quadrant_counts[1] + quadrant_counts[3];
    n_second_low = quadrant_counts[0] + quadrant_counts[2];

    n_total = n_first_low + n_first_high;

    for (int impl_type = 0; impl_type < 4; impl_type++){
        net->getSingleImplication(quadrant_counts, n_total, n_first_low, n_first_high, n_second_low, n_second_high, impl_type, &statistic, &pval);
        if (statistic > statThresh && pval < pvalThresh){
            printf("%ld\t%ld\t%d\t%f\t%f\t\n", gene1, gene2, impl_type, statistic, pval);
        }
    }
}

